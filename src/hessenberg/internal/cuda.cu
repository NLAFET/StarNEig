#include "hip/hip_runtime.h"
///
/// @file
///
/// @author Mirko Myllykoski (mirkom@cs.umu.se), Umeå University
///
/// @internal LICENSE
///
/// Copyright (c) 2019-2020, Umeå Universitet
///
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions are met:
///
/// 1. Redistributions of source code must retain the above copyright notice,
///    this list of conditions and the following disclaimer.
///
/// 2. Redistributions in binary form must reproduce the above copyright notice,
///    this list of conditions and the following disclaimer in the documentation
///    and/or other materials provided with the distribution.
///
/// 3. Neither the name of the copyright holder nor the names of its
///    contributors may be used to endorse or promote products derived from this
///    software without specific prior written permission.
///
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
/// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
/// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
/// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
/// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
/// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
/// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
/// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
/// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
/// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.
///

#include <starneig_config.h>
#include <starneig/configuration.h>
#include "hip/hip_runtime.h"
#include "../common/common.h"
#include "../common/tiles.h"
#include <starpu.h>
#include <starpu_cublas_v2.h>

static const double *one = (const double[]) { 1.0 };
static const double *m_one = (const double[]) { -1.0 };
static const double *zero = (const double[]) { 0.0 };

extern "C" void dlarfg_(int const *, double *, double *, int const *, double *);

///
/// @brief Custom matrix-vector multiplication CUDA kernel.
///
/// @param[in]  rbegin  first row that is included to the computation
/// @param[in]  rend    last row that is included to the computation + 1
/// @param[in]  cbegin  first column that is included to the computation
/// @param[in]  cend    last column that is included to the computation + 1
/// @param[in]  bm      tile height
/// @param[in]  bn      tile width
/// @param[in]  A       device side argument buffer (matrix tiles)
/// @param[in]  x       device side argument buffer (input vector)
/// @param[out] y       device side argument buffer (output vector)
///
static __global__ void tiled_matrix_vector(
    int rbegin, int rend, int cbegin, int cend, int bm, int bn,
    struct tile_addr const * __restrict__ A, uintptr_t const * __restrict__ x,
    uintptr_t * __restrict__ y)
{
    extern __shared__ double s[];

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int tid = idx / bm;                         // tile row index
    int rid = idx % bm;                         // row index inside the tile row
    int rtiles = (rend-1)/bm + 1 - rbegin/bm;   // tile row count

    double v = 0.0;

    if (rbegin <= idx && idx < rend) {

        // loop over the tile columns
        int cbbegin = cbegin/bn;
        int cbend = (cend-1)/bn + 1;
        for (int i = cbbegin; i < cbend; i++) {

            // compute the correct row address inside the tile
            double const * __restrict__ ptr =
                (double const *) A[i*rtiles+tid].ptr;
            int ld = A[i*rtiles+tid].ld;

            // loop over the columns in the tile (blockDim.y threads per row)
            int begin = MAX(0, cbegin - i*bn);
            int end = MIN(bn, cend - i*bn);
            for (int j = begin+threadIdx.y; j < end; j += blockDim.y)
                v += ptr[j*ld+rid] * ((double const *) x[i])[j];
        }
    }

    // store partial sums to the shared memory
    if (0 < threadIdx.y && rbegin <= idx && idx < rend)
        s[(threadIdx.y-1)*blockDim.x+threadIdx.x] = v;
    __syncthreads();

    // sum partial sums together and store the final result
    if (threadIdx.y == 0 && rbegin <= idx && idx < rend) {
        for (int i = 0; i < blockDim.y-1; i++)
            v += s[i*blockDim.x+threadIdx.x];
        ((double *)y[tid])[rid] = v;
    }
}

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////

void starneig_hessenberg_cuda_compute_column(
    void *buffers[], void *cl_args)
{
    struct packing_info A_pi;
    struct range_packing_info v_pi, y_pi;
    starpu_codelet_unpack_args(cl_args, &A_pi, &v_pi, &y_pi);

    int k = 0;

    // involved trailing matrix tiles
    struct tile_addr *A_da =
        starneig_cuda_prepare_join_window(&A_pi, buffers + k);
    k += A_pi.handles;

    // intemediate vector interface for the trailing matrix operation
    uintptr_t *v_da = starneig_cuda_prepare_join_range(&v_pi, buffers + k);
    k += v_pi.handles;

    // intemediate vector interface from the trailing matrix operation
    uintptr_t *y_da = starneig_cuda_prepare_join_range(&y_pi, buffers + k);
    k += y_pi.handles;

    hipStream_t stream = starpu_cuda_get_local_stream();

    int rtiles = (A_pi.rend-1)/A_pi.bm + 1 - A_pi.rbegin/A_pi.bm;

    dim3 threads(32, MIN(32, MAX(1, (A_pi.cend-A_pi.cbegin)/16)));
    dim3 blocks(divceil(rtiles*A_pi.bm, threads.x));
    size_t shared_size = threads.x*(threads.y-1)*sizeof(double);

    tiled_matrix_vector<<<blocks, threads, shared_size, stream>>>(
        A_pi.rbegin, A_pi.rend, A_pi.cbegin, A_pi.cend, A_pi.bm, A_pi.bn,
        A_da, v_da, y_da);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        STARPU_CUDA_REPORT_ERROR(err);
}

extern "C" void starneig_hessenberg_cuda_update_trail(
    void *buffers[], void *cl_args)
{
    struct packing_info packing_info;
    int nb, offset;
    starpu_codelet_unpack_args(cl_args, &packing_info, &nb, &offset);

    int m = packing_info.rend - packing_info.rbegin;
    int n = packing_info.cend - packing_info.cbegin;

    double *V = (double *) STARPU_MATRIX_GET_PTR(buffers[0]);
    int ldV = STARPU_MATRIX_GET_LD(buffers[0]);

    double *T = (double *) STARPU_MATRIX_GET_PTR(buffers[1]);
    int ldT = STARPU_MATRIX_GET_LD(buffers[1]);

    double *Y = (double *) STARPU_MATRIX_GET_PTR(buffers[2]);
    int ldY = STARPU_MATRIX_GET_LD(buffers[2]);

    double *A = (double *) STARPU_MATRIX_GET_PTR(buffers[3]);
    int nA = STARPU_MATRIX_GET_NY(buffers[3]);
    int ldA = STARPU_MATRIX_GET_LD(buffers[3]);

    double *W = (double *) STARPU_MATRIX_GET_PTR(buffers[4]);
    int mW = STARPU_MATRIX_GET_NX(buffers[4]);
    int ldW = STARPU_MATRIX_GET_LD(buffers[4]);

    int max_width = MIN(nA, mW);

    struct tile_addr *device_args =
        starneig_cuda_prepare_join_window(&packing_info, buffers+5);

    hipStream_t stream = starpu_cuda_get_local_stream();
    hipblasHandle_t handle = starpu_cublas_get_local_handle();
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
    hipblasSetStream(handle, stream);

    for (int i = 0; i < n; i += max_width) {

        //
        // join tiles and update from the right
        //

        starneig_cuda_join_sub_window(0, m, i, MIN(n, i+max_width),
            stream, &packing_info, device_args, ldA, A, 0);

        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
            m, MIN(max_width, n-i), nb, m_one,
            Y, ldY, V+offset+i+nb-1, ldV, one, A, ldA);

        //
        // update from the left
        //

        int width = MIN(max_width, n-i);
        if (0 < width) {
            hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, width, nb,
                one, A, ldA, zero, A, ldA, W, ldW);

            hipblasDtrmm(
                handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_UNIT, width, nb, one, V, ldV, W, ldW, W, ldW);

            if (nb < m)
                hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, width, nb, m-nb,
                    one, A+nb, ldA, V+nb, ldV, one, W, ldW);

            hipblasDtrmm(
                handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
                HIPBLAS_DIAG_NON_UNIT, width, nb, one, T, ldT, W, ldW, W, ldW);

            if (nb < m)
                hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m-nb, width, nb,
                    m_one, V+nb, ldV, W, ldW, one, A+nb, ldA);

            hipblasDtrmm(
                handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T,
                HIPBLAS_DIAG_UNIT, width, nb, one, V, ldV, W, ldW, W, ldW);

            hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nb, width,
                one, A, ldA, m_one, W, ldW, A, ldA);
        }

        //
        // copy tiles back
        //

        starneig_cuda_join_sub_window(0, m, i, MIN(n, i+max_width),
            stream, &packing_info, device_args, ldA, A, 1);
    }
}

extern "C" void starneig_hessenberg_cuda_update_right(
    void *buffers[], void *cl_args)
{
    struct packing_info packing_info;
    int nb;
    starpu_codelet_unpack_args(cl_args, &packing_info, &nb);

    int m = packing_info.rend - packing_info.rbegin;
    int n = packing_info.cend - packing_info.cbegin;

    double *V = (double *) STARPU_MATRIX_GET_PTR(buffers[0]);
    int ldV = STARPU_MATRIX_GET_LD(buffers[0]);

    double *T = (double *) STARPU_MATRIX_GET_PTR(buffers[1]);
    int ldT = STARPU_MATRIX_GET_LD(buffers[1]);

    double *A = (double *) STARPU_MATRIX_GET_PTR(buffers[2]);
    int ldA = STARPU_MATRIX_GET_LD(buffers[2]);

    double *W = (double *) STARPU_MATRIX_GET_PTR(buffers[3]);
    int ldW = STARPU_MATRIX_GET_LD(buffers[3]);

    struct tile_addr *device_args =
        starneig_cuda_prepare_join_window(&packing_info, buffers + 4);

    hipStream_t stream = starpu_cuda_get_local_stream();
    hipblasHandle_t handle = starpu_cublas_get_local_handle();
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
    hipblasSetStream(handle, stream);

    starneig_cuda_join_window(stream, &packing_info, device_args, ldA, A, 0);

    hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, nb,
        one, A, ldA, zero, A, ldA, W, ldW);

    hipblasDtrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, m, nb, one, V, ldV, W, ldW, W, ldW);

    if (nb < n)
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, nb, n-nb,
            one, A+nb*ldA, ldA, V+nb, ldV, one, W, ldW);

    hipblasDtrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER,
        HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, nb, one, T, ldT, W, ldW, W, ldW);

    if (nb < n)
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n-nb, nb,
            m_one, W, ldW, V+nb, ldV, one, A+nb*ldA, ldA);

    hipblasDtrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_T, HIPBLAS_DIAG_UNIT, m, nb, one, V, ldV, W, ldW, W, ldW);

    hipblasDgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, nb,
        one, A, ldA, m_one, W, ldW, A, ldA);

    starneig_cuda_join_window(stream, &packing_info, device_args, ldA, A, 1);
}

extern "C" void starneig_hessenberg_cuda_update_left(
    void *buffers[], void *cl_args)
{
    struct packing_info packing_info;
    int nb;
    starpu_codelet_unpack_args(cl_args, &packing_info, &nb);

    int m = packing_info.rend - packing_info.rbegin;
    int n = packing_info.cend - packing_info.cbegin;

    double *V = (double *) STARPU_MATRIX_GET_PTR(buffers[0]);
    int ldV = STARPU_MATRIX_GET_LD(buffers[0]);

    double *T = (double *) STARPU_MATRIX_GET_PTR(buffers[1]);
    int ldT = STARPU_MATRIX_GET_LD(buffers[1]);

    double *A = (double *) STARPU_MATRIX_GET_PTR(buffers[2]);
    int ldA = STARPU_MATRIX_GET_LD(buffers[2]);

    double *W = (double *) STARPU_MATRIX_GET_PTR(buffers[3]);
    int ldW = STARPU_MATRIX_GET_LD(buffers[3]);

    struct tile_addr *device_args =
        starneig_cuda_prepare_join_window(&packing_info, buffers+4);

    hipStream_t stream = starpu_cuda_get_local_stream();
    hipblasHandle_t handle = starpu_cublas_get_local_handle();
    hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
    hipblasSetStream(handle, stream);

    starneig_cuda_join_window(stream, &packing_info, device_args, ldA, A, 0);

    hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, nb,
        one, A, ldA, zero, A, ldA, W, ldW);

    hipblasDtrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, nb, one, V, ldV, W, ldW, W, ldW);

    if (nb < m)
        hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, nb, m-nb,
            one, A+nb, ldA, V+nb, ldV, one, W, ldW);

    hipblasDtrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER,
        HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, nb, one, T, ldT, W, ldW, W, ldW);

    if (nb < m)
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m-nb, n, nb,
            m_one, V+nb, ldV, W, ldW, one, A+nb, ldA);

    hipblasDtrmm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
        HIPBLAS_OP_T, HIPBLAS_DIAG_UNIT, n, nb, one, V, ldV, W, ldW, W, ldW);

    hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nb, n,
        m_one, W, ldW, one, A, ldA, A, ldA);

    starneig_cuda_join_window(stream, &packing_info, device_args, ldA, A, 1);
}
